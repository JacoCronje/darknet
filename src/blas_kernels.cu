#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"
#include <assert.h>

extern "C" {
#include "blas.h"
#include "hip/hip_runtime.h"
#include "utils.h"
}

__global__ void normalize_kernel(int N, float *x, float *mean, float *variance, int batch, int filters, int spatial)
{
    int index = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (index >= N) return;
    int f = (index/spatial)%filters;
    
    x[index] = (x[index] - mean[f])/(sqrt(variance[f]) + .000001f);
}

__global__ void normalize_delta_kernel(int N, float *x, float *mean, float *variance, float *mean_delta, float *variance_delta, int batch, int filters, int spatial, float *delta)
{
    int index = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (index >= N) return;
    int f = (index/spatial)%filters;
    
    delta[index] = delta[index] * 1./(sqrt(variance[f]) + .000001f) + variance_delta[f] * 2. * (x[index] - mean[f]) / (spatial * batch) + mean_delta[f]/(spatial*batch);
}

extern "C" void normalize_delta_gpu(float *x, float *mean, float *variance, float *mean_delta, float *variance_delta, int batch, int filters, int spatial, float *delta)
{
    size_t N = batch*filters*spatial;
    normalize_delta_kernel<<<cuda_gridsize(N), BLOCK>>>(N, x, mean, variance, mean_delta, variance_delta, batch, filters, spatial, delta);
    check_error(hipPeekAtLastError());
}

__global__ void  variance_delta_kernel(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= filters) return;
    int j,k;
    variance_delta[i] = 0;
    for(j = 0; j < batch; ++j){
        for(k = 0; k < spatial; ++k){
            int index = j*filters*spatial + i*spatial + k;
            variance_delta[i] += delta[index]*(x[index] - mean[i]);
        }
    }
    variance_delta[i] *= -.5 * pow(variance[i] + .000001f, (float)(-3./2.));
}

__global__ void accumulate_kernel(float *x, int n, int groups, float *sum)
{
    int k;
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= groups) return;
    sum[i] = 0;
    for(k = 0; k < n; ++k){
        sum[i] += x[k*groups + i];
    }
}

__global__ void fast_mean_delta_kernel(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    local[id] = 0;

    int filter = blockIdx.x;

    int i, j;
    for(j = 0; j < batch; ++j){
        for(i = 0; i < spatial; i += threads){
            int index = j*spatial*filters + filter*spatial + i + id;
            local[id] += (i+id < spatial) ? delta[index] : 0;
        }
    }

    if(id == 0){
        mean_delta[filter] = 0;
        for(i = 0; i < threads; ++i){
            mean_delta[filter] += local[i];
        }
        mean_delta[filter] *= (-1./sqrt(variance[filter] + .000001f));
    }
}

__global__ void  fast_variance_delta_kernel(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    local[id] = 0;

    int filter = blockIdx.x;

    int i, j;
    for(j = 0; j < batch; ++j){
        for(i = 0; i < spatial; i += threads){
            int index = j*spatial*filters + filter*spatial + i + id;

            local[id] += (i+id < spatial) ? delta[index]*(x[index] - mean[filter]) : 0;
        }
    }

    if(id == 0){
        variance_delta[filter] = 0;
        for(i = 0; i < threads; ++i){
            variance_delta[filter] += local[i];
        }
        variance_delta[filter] *= -.5 * pow(variance[filter] + .000001f, (float)(-3./2.));
    }
}


__global__ void mean_delta_kernel(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= filters) return;
    int j,k;
    mean_delta[i] = 0;
    for (j = 0; j < batch; ++j) {
        for (k = 0; k < spatial; ++k) {
            int index = j*filters*spatial + i*spatial + k;
            mean_delta[i] += delta[index];
        }
    }
    mean_delta[i] *= (-1./sqrt(variance[i] + .000001f));
}

extern "C" void mean_delta_gpu(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    mean_delta_kernel<<<cuda_gridsize(filters), BLOCK>>>(delta, variance, batch, filters, spatial, mean_delta);
    check_error(hipPeekAtLastError());
}

extern "C" void fast_mean_delta_gpu(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    fast_mean_delta_kernel<<<filters, BLOCK>>>(delta, variance, batch, filters, spatial, mean_delta);
    check_error(hipPeekAtLastError());
}

extern "C" void fast_variance_delta_gpu(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta)
{
    fast_variance_delta_kernel<<<filters, BLOCK>>>(x, delta, mean, variance, batch, filters, spatial, variance_delta);
    check_error(hipPeekAtLastError());
}

__global__ void  mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
    float scale = 1./(batch * spatial);
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= filters) return;
    int j,k;
    mean[i] = 0;
    for(j = 0; j < batch; ++j){
        for(k = 0; k < spatial; ++k){
            int index = j*filters*spatial + i*spatial + k;
            mean[i] += x[index];
        }
    }
    mean[i] *= scale;
}

__global__ void variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    float scale = 1./(batch * spatial - 1);
    int j,k;
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= filters) return;
    variance[i] = 0;
    for(j = 0; j < batch; ++j){
        for(k = 0; k < spatial; ++k){
            int index = j*filters*spatial + i*spatial + k;
            variance[i] += pow((x[index] - mean[i]), 2);
        }
    }
    variance[i] *= scale;
}

__global__ void axpy_kernel(int N, float ALPHA, float *X, int OFFX, int INCX,  float *Y, int OFFY, int INCY)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) Y[OFFY+i*INCY] += ALPHA*X[OFFX+i*INCX];
}

__global__ void pow_kernel(int N, float ALPHA, float *X, int INCX, float *Y, int INCY)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) Y[i*INCY] = pow(X[i*INCX], ALPHA);
}

__global__ void const_kernel(int N, float ALPHA, float *X, int INCX)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) X[i*INCX] = ALPHA;
}

__global__ void scal_kernel(int N, float ALPHA, float *X, int INCX)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) X[i*INCX] *= ALPHA;
}

__global__ void fill_kernel(int N, float ALPHA, float *X, int INCX)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) X[i*INCX] = ALPHA;
}

__global__ void mask_kernel(int n,  float *x, float mask_num, float *mask)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < n && mask[i] == mask_num) x[i] = mask_num;
}

__global__ void copy_kernel(int N,  float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) Y[i*INCY + OFFY] = X[i*INCX + OFFX];
}

__global__ void mul_kernel(int N, float *X, int INCX, float *Y, int INCY)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) Y[i*INCY] *= X[i*INCX];
}


extern "C" void normalize_gpu(float *x, float *mean, float *variance, int batch, int filters, int spatial)
{
    size_t N = batch*filters*spatial;
    normalize_kernel<<<cuda_gridsize(N), BLOCK>>>(N, x, mean, variance, batch, filters, spatial);
    check_error(hipPeekAtLastError());
}

__global__ void  fast_mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    local[id] = 0;

    int filter = blockIdx.x;

    int i, j;
    for(j = 0; j < batch; ++j){
        for(i = 0; i < spatial; i += threads){
            int index = j*spatial*filters + filter*spatial + i + id;
            local[id] += (i+id < spatial) ? x[index] : 0;
        }
    }

    if(id == 0){
        mean[filter] = 0;
        for(i = 0; i < threads; ++i){
            mean[filter] += local[i];
        }
        mean[filter] /= spatial * batch;
    }
}

__global__ void  fast_variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    local[id] = 0;

    int filter = blockIdx.x;

    int i, j;
    for(j = 0; j < batch; ++j){
        for(i = 0; i < spatial; i += threads){
            int index = j*spatial*filters + filter*spatial + i + id;

            local[id] += (i+id < spatial) ? pow((x[index] - mean[filter]), 2) : 0;
        }
    }

    if(id == 0){
        variance[filter] = 0;
        for(i = 0; i < threads; ++i){
            variance[filter] += local[i];
        }
        variance[filter] /= (spatial * batch - 1);
    }
}

extern "C" void fast_mean_gpu(float *x, int batch, int filters, int spatial, float *mean)
{
    fast_mean_kernel<<<filters, BLOCK>>>(x, batch, filters, spatial, mean);
    check_error(hipPeekAtLastError());
}

extern "C" void fast_variance_gpu(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    fast_variance_kernel<<<filters, BLOCK>>>(x, mean, batch, filters, spatial, variance);
    check_error(hipPeekAtLastError());
}


extern "C" void mean_gpu(float *x, int batch, int filters, int spatial, float *mean)
{
    mean_kernel<<<cuda_gridsize(filters), BLOCK>>>(x, batch, filters, spatial, mean);
    check_error(hipPeekAtLastError());
}

extern "C" void variance_gpu(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    variance_kernel<<<cuda_gridsize(filters), BLOCK>>>(x, mean, batch, filters, spatial, variance);
    check_error(hipPeekAtLastError());
}

extern "C" void axpy_ongpu(int N, float ALPHA, float * X, int INCX, float * Y, int INCY)
{
    axpy_ongpu_offset(N, ALPHA, X, 0, INCX, Y, 0, INCY);
}

extern "C" void pow_ongpu(int N, float ALPHA, float * X, int INCX, float * Y, int INCY)
{
    pow_kernel<<<cuda_gridsize(N), BLOCK>>>(N, ALPHA, X, INCX, Y, INCY);
    check_error(hipPeekAtLastError());
}

extern "C" void axpy_ongpu_offset(int N, float ALPHA, float * X, int OFFX, int INCX, float * Y, int OFFY, int INCY)
{
    axpy_kernel<<<cuda_gridsize(N), BLOCK>>>(N, ALPHA, X, OFFX, INCX, Y, OFFY, INCY);
    check_error(hipPeekAtLastError());
}

extern "C" void copy_ongpu(int N, float * X, int INCX, float * Y, int INCY)
{
    copy_ongpu_offset(N, X, 0, INCX, Y, 0, INCY);
}

extern "C" void mul_ongpu(int N, float * X, int INCX, float * Y, int INCY)
{
    mul_kernel<<<cuda_gridsize(N), BLOCK>>>(N, X, INCX, Y, INCY);
    check_error(hipPeekAtLastError());
}

extern "C" void copy_ongpu_offset(int N, float * X, int OFFX, int INCX, float * Y, int OFFY, int INCY)
{
    copy_kernel<<<cuda_gridsize(N), BLOCK>>>(N, X, OFFX, INCX, Y, OFFY, INCY);
    check_error(hipPeekAtLastError());
}

extern "C" void mask_ongpu(int N, float * X, float mask_num, float * mask)
{
    mask_kernel<<<cuda_gridsize(N), BLOCK>>>(N, X, mask_num, mask);
    check_error(hipPeekAtLastError());
}

extern "C" void const_ongpu(int N, float ALPHA, float * X, int INCX)
{
    const_kernel<<<cuda_gridsize(N), BLOCK>>>(N, ALPHA, X, INCX);
    check_error(hipPeekAtLastError());
}

extern "C" void scal_ongpu(int N, float ALPHA, float * X, int INCX)
{
    scal_kernel<<<cuda_gridsize(N), BLOCK>>>(N, ALPHA, X, INCX);
    check_error(hipPeekAtLastError());
}

extern "C" void fill_ongpu(int N, float ALPHA, float * X, int INCX)
{
    fill_kernel<<<cuda_gridsize(N), BLOCK>>>(N, ALPHA, X, INCX);
    check_error(hipPeekAtLastError());
}

__global__ void shortcut_kernel(int size, int minw, int minh, int minc, int stride, int sample, int batch, int w1, int h1, int c1, float *add, int w2, int h2, int c2, float *out)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= size) return;
    int i = id % minw;
    id /= minw;
    int j = id % minh;
    id /= minh;
    int k = id % minc;
    id /= minc;
    int b = id % batch;

    int out_index = i*sample + w2*(j*sample + h2*(k + c2*b));
    int add_index = i*stride + w1*(j*stride + h1*(k + c1*b));
    out[out_index] += add[add_index];
}

extern "C" void shortcut_gpu(int batch, int w1, int h1, int c1, float *add, int w2, int h2, int c2, float *out)
{
    int minw = (w1 < w2) ? w1 : w2;
    int minh = (h1 < h2) ? h1 : h2;
    int minc = (c1 < c2) ? c1 : c2;

    int stride = w1/w2;
    int sample = w2/w1;
    assert(stride == h1/h2);
    assert(sample == h2/h1);
    if(stride < 1) stride = 1;
    if(sample < 1) sample = 1;

    int size = batch * minw * minh * minc;
    shortcut_kernel<<<cuda_gridsize(size), BLOCK>>>(size, minw, minh, minc, stride, sample, batch, w1, h1, c1, add, w2, h2, c2, out);
    check_error(hipPeekAtLastError());
}

__global__ void smooth_l1_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < n){
        float diff = truth[i] - pred[i];
        float abs_val = abs(diff);
        if(abs_val < 1) {
            error[i] = diff * diff;
            delta[i] = diff;
        }
        else {
            error[i] = 2*abs_val - 1;
            delta[i] = (diff < 0) ? -1 : 1;
        }
    }
}

extern "C" void smooth_l1_gpu(int n, float *pred, float *truth, float *delta, float *error)
{
    smooth_l1_kernel<<<cuda_gridsize(n), BLOCK>>>(n, pred, truth, delta, error);
    check_error(hipPeekAtLastError());
}

__global__ void l2_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < n){
        float diff = truth[i] - pred[i];
        error[i] = diff * diff; //I know this is technically wrong, deal with it.
        delta[i] = diff;
    }
}

extern "C" void l2_gpu(int n, float *pred, float *truth, float *delta, float *error)
{
    l2_kernel<<<cuda_gridsize(n), BLOCK>>>(n, pred, truth, delta, error);
    check_error(hipPeekAtLastError());
}

__global__ void routescale_kernel(int size, int w1, int h1, int c1, float *src, int w2, int h2, int c2, float *out)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= size) return;
    int x = id % w2;
    id /= w2;
    int y = id % h2;
    id /= h2;
    int c = id % c1;

    // no interpolation
//    int src_index = (x*w1/w2) + w1*(y*h1/h2) + c*w1*h1;
 //   int out_index = x + w2*y + c*w2*h2;
  //  out[out_index] = src[src_index];

    // interpolation
    int x1 = x*w1/w2;
    int y1 = y*h1/h2;
    float x1p = (float)(x1*w2)/w1;
    float y1p = (float)(y1*h2)/h1;
    float x2p = (float)((x1+1)*w2)/w1;
    float y2p = (float)((y1+1)*h2)/h1;
    float xDelta = ((float)(x) - x1p) / (x2p-x1p);
    float yDelta = ((float)(y) - y1p) / (y2p-y1p);

    // read corner values
    int ridx = x1 + w1*y1 + c*w1*h1;
    float pa = src[ridx];
    float pb = src[ridx+1];
    float pc = src[ridx+1+w1];
    float pd = src[ridx+w1];

    float ab = (1.f-xDelta)*pa + (xDelta)*pb;
    float dc = (1.f-xDelta)*pd + (xDelta)*pc;
    float f = (1.f-yDelta)*ab + (yDelta)*dc;
    int out_index = x + w2*y + c*w2*h2;
    out[out_index] = f;


}

extern "C" void routescale_gpu(int w1, int h1, int c1, float *src, int w2, int h2, int c2, float *out)
{
    int size = w2*h2*c1;
    routescale_kernel<<<cuda_gridsize(size), BLOCK>>>(size, w1, h1, c1, src, w2, h2, c2, out);
    check_error(hipPeekAtLastError());
}

__global__ void routedelta_kernel(int size, int w1, int h1, int c1, float *delta, int w2, int h2, int c2, float *src)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= size) return;
    int x = id % w2;
    id /= w2;
    int y = id % h2;
    id /= h2;
    int c = id % c1;

    // no interpolation
//    int out_index = (x*w1/w2) + w1*(y*h1/h2) + c*w1*h1;
//    int src_index = x + w2*y + c*w2*h2;
//   // float ratio = (float)(w1*h1) / (w2*h2);
//    float f = src[src_index];// * ratio;
//    delta += out_index;
//    atomicAdd(delta, f);

    // interpolation
    int x1 = x*w1/w2;
    int y1 = y*h1/h2;
    float x1p = (float)(x1*w2)/w1;
    float y1p = (float)(y1*h2)/h1;
    float x2p = (float)((x1+1)*w2)/w1;
    float y2p = (float)((y1+1)*h2)/h1;
    float xDelta = ((float)(x) - x1p) / (x2p-x1p);
    float yDelta = ((float)(y) - y1p) / (y2p-y1p);

    int out_index = x1 + w1*y1 + c*w1*h1;
    int src_index = x + w2*y + c*w2*h2;
    float f = src[src_index];
    delta += out_index;
    float fa = f * (1.f - xDelta) * (1.f - yDelta);
    float fb = f * (xDelta) * (1.f - yDelta);
    float fc = f * (xDelta) * (yDelta);
    float fd = f * (1.f - xDelta) * (yDelta);
    atomicAdd(delta, fa);
    delta += 1;
    atomicAdd(delta, fb);
    delta += w1;
    atomicAdd(delta, fc);
    delta -= 1;
    atomicAdd(delta, fd);
}

extern "C" void routedelta_gpu(int w1, int h1, int c1, float *delta, int w2, int h2, int c2, float *src)
{
    int size = w2*h2*c1;
    routedelta_kernel<<<cuda_gridsize(size), BLOCK>>>(size, w1, h1, c1, delta, w2, h2, c2, src);
    check_error(hipPeekAtLastError());
}



__global__ void augmentflip_kernel(int size, int w, int h, float *src, float *out)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= size) return;
    int x = id % w;
    id /= w;
    int y = id % h;

    float f = src[x+y*w];
    int out_index = (w-x-1)+y*w;
    out[out_index] = f;
}

__global__ void augmentflip_delta_kernel(int size, int w, int h, float ALPHA, float *src, float *out)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= size) return;
    int x = id % w;
    id /= w;
    int y = id % h;

    float f = src[x+y*w] * ALPHA;
    int out_index = (w-x-1)+y*w;
    out[out_index] += f;
}

extern "C" void augmentflip_gpu(int w, int h, float *src, float *dest)
{
    int size = w*h;
    augmentflip_kernel<<<cuda_gridsize(size), BLOCK>>>(size, w, h, src, dest);
    check_error(hipPeekAtLastError());
}

extern "C" void augmentflip_delta_gpu(int w, int h, float ALPHA, float *src, float *dest)
{
    int size = w*h;
    augmentflip_delta_kernel<<<cuda_gridsize(size), BLOCK>>>(size, w, h, ALPHA, src, dest);
    check_error(hipPeekAtLastError());
}




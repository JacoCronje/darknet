#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"

extern "C" {
#include "augment_layer.h"
#include "hip/hip_runtime.h"
}

__global__ void augmentflip_kernel(int size, int w, int h, float *src, float *out)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= size) return;
    int x = id % w;
    id /= w;
    int y = id % h;

    float f = src[x+y*w];
    int out_index = (w-x-1)+y*w;
    out[out_index] = f;
}

__global__ void augmentflip_delta_kernel(int size, int w, int h, float ALPHA, float *src, float *out)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= size) return;
    int x = id % w;
    id /= w;
    int y = id % h;

    float f = src[x+y*w] * ALPHA;
    int out_index = (w-x-1)+y*w;
    out[out_index] += f;
}

extern "C" void augmentflip_gpu(int w, int h, float *src, float *dest)
{
    int size = w*h;
    augmentflip_kernel<<<cuda_gridsize(size), BLOCK>>>(size, w, h, src, dest);
    check_error(hipPeekAtLastError());
}

extern "C" void augmentflip_delta_gpu(int w, int h, float ALPHA, float *src, float *dest)
{
    int size = w*h;
    augmentflip_delta_kernel<<<cuda_gridsize(size), BLOCK>>>(size, w, h, ALPHA, src, dest);
    check_error(hipPeekAtLastError());
}


__device__ float get_pixel_kernel(float *image, int w, int h, int x, int y)
{
    if(x < 0 || x >= w || y < 0 || y >= h) return 0;
    return image[x + w*y];
}

__device__ float bilinear_interpolate_kernel(float *image, int w, int h, float x, float y)
{
    int ix = (int) floorf(x);
    int iy = (int) floorf(y);

    float dx = x - ix;
    float dy = y - iy;

    float val = (1-dy) * (1-dx) * get_pixel_kernel(image, w, h, ix, iy) +
        dy     * (1-dx) * get_pixel_kernel(image, w, h, ix, iy+1) +
        (1-dy) *   dx   * get_pixel_kernel(image, w, h, ix+1, iy) +
        dy     *   dx   * get_pixel_kernel(image, w, h, ix+1, iy+1);
    return val;
}


__global__ void augmentrotate_kernel(int size, int w, int h, float *src, float *out, float angle)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= size) return;
    int x = id % w;
    id /= w;
    int y = id % h;

    int cx = w/2;
    int cy = h/2;

    float rx = cos(angle)*(x-cx) - sin(angle)*(y-cy) + cx;
    float ry = sin(angle)*(x-cx) + cos(angle)*(y-cy) + cy;
    int out_index = x+y*w;
    out[out_index] = bilinear_interpolate_kernel(src, w, h, rx, ry);
}

__global__ void augmentrotate_delta_kernel(int size, int w, int h, float ALPHA, float *src, float *out, float angle)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= size) return;
    int x = id % w;
    id /= w;
    int y = id % h;

    int cx = w/2;
    int cy = h/2;

    float rx = cos(angle)*(x-cx) - sin(angle)*(y-cy) + cx;
    float ry = sin(angle)*(x-cx) + cos(angle)*(y-cy) + cy;
    int out_index = x+y*w;
    out[out_index] += ALPHA*bilinear_interpolate_kernel(src, w, h, rx, ry);

}

extern "C" void augmentrotate_gpu(int w, int h, float *src, float *dest, int ang)
{
    float radians = (float)(ang)*3.14159265/180.;
    int size = w*h;
    augmentrotate_kernel<<<cuda_gridsize(size), BLOCK>>>(size, w, h, src, dest, radians);
    check_error(hipPeekAtLastError());
}

extern "C" void augmentrotate_delta_gpu(int w, int h, float ALPHA, float *src, float *dest, int ang)
{
    float radians = -(float)(ang)*3.14159265/180.;
    int size = w*h;
    augmentrotate_delta_kernel<<<cuda_gridsize(size), BLOCK>>>(size, w, h, ALPHA, src, dest, radians);
    check_error(hipPeekAtLastError());
}





__global__ void forward_augment_flip_kernel(int n, int out_w, int out_h, int out_c,
                                               int gap, int w, int h,
                                               float *src, float *dest)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(id >= n) return;

    int x = id % out_w;
    id /= out_w;
    int y = id % out_h;
    id /= out_h;
    int c = id % out_c;
    id /= out_c;
    int b = id;

    int widx = x + out_w*(y + out_h*(c + b*out_c));
    float val = 0;
    if (y<h)
    {
        // copy
        int ridx = x + w*(y + h*(c + b*out_c));
        val = src[ridx];
    } else if (y>=h+gap)
    {
        // flip
        int ridx = (w-x-1) + w*((y-gap-h) + h*(c + b*out_c));
        val = src[ridx];
    }
    dest[widx] = val;
}

__global__ void backward_augment_flip_kernel(int n, int out_w, int out_h, int out_c,
                                             int gap, int w, int h,
                                             float *src, float *dest)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(id >= n) return;

    int x = id % w;
    id /= w;
    int y = id % h;
    id /= h;
    int c = id % out_c;
    id /= out_c;
    int b = id;

    int widx = x + y*w + c*w*h + b*out_c*w*h;
    int ridx = x + y*out_w + c*out_w*out_h + b*out_w*out_h*out_c;
    float dt = 0.5 * src[ridx];
    ridx = (w-x-1) + (y+gap+h)*out_w + c*out_w*out_h + b*out_w*out_h*out_c;
    dt += 0.5 * src[ridx];
    dest[widx] += dt;
}

extern "C" void augment_flip_gpu(int w, int h, int c, int batch, int gap,
                                 float *src, float *dest)
{
    int out_w = w;
    int out_h = 2*h + gap;
    int out_c = c;
    size_t n = out_h*out_w*out_c*batch;

    forward_augment_flip_kernel<<<cuda_gridsize(n), BLOCK>>>(n, out_w, out_h, out_c, gap, w, h,
                                                             src, dest);
    check_error(hipPeekAtLastError());
}

extern "C" void augment_flip_delta_gpu(int w, int h, int c, int batch, int gap,
                                       float *src, float *dest)
{
    int out_w = w;
    int out_h = 2*h + gap;
    int out_c = c;
    size_t n = h*w*c*batch;

    backward_augment_flip_kernel<<<cuda_gridsize(n), BLOCK>>>(n, out_w, out_h, out_c, gap, w, h,
                                                             src, dest);
    check_error(hipPeekAtLastError());
}



__global__ void augment_forward_kernel(int size, int w, int h, float *src, float *out, float angle, int flip, float scale)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= size) return;
    int x = id % w;
    id /= w;
    int y = id % h;

    int cx = w/2;
    int cy = h/2;

    float rx = scale*(cos(angle)*(x-cx) - sin(angle)*(y-cy)) + cx;
    float ry = scale*(sin(angle)*(x-cx) + cos(angle)*(y-cy)) + cy;
    rx = (flip ? (w-rx-1) : rx);
    int out_index = x+y*w;//(flip ? (w-x-1) : x) + y*w;

    int ix = (int)floorf(rx);
    int iy = (int)floorf(ry);
    float val = 0;
    if (!(ix < 0 || ix >= w || iy < 0 || iy >= h))
    {
        val = src[ix+iy*w];
    }
    out[out_index] = val;//bilinear_interpolate_kernel(src, w, h, rx, ry);
}


extern "C" void augment_forward_gpu(int w, int h, float *src, float *dest, float angle, int flip, float scale)
{
    float radians = (float)(angle)*3.14159265/180.;
    int size = w*h;
    augment_forward_kernel<<<cuda_gridsize(size), BLOCK>>>(size, w, h, src, dest, radians, flip, scale);
    check_error(hipPeekAtLastError());
}

__global__ void augment_backward_kernel(int size, int w, int h, float *src, float *out, float angle, int flip, float scale, float ALPHA)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= size) return;
    int x = id % w;
    id /= w;
    int y = id % h;

    int cx = w/2;
    int cy = h/2;

    float rx = scale*(cos(angle)*(x-cx) - sin(angle)*(y-cy)) + cx;
    float ry = scale*(sin(angle)*(x-cx) + cos(angle)*(y-cy)) + cy;
    rx = (flip ? (w-rx-1) : rx);

    int ix = (int)floorf(rx);
    int iy = (int)floorf(ry);
    if ((ix < 0 || ix >= w || iy < 0 || iy >= h)) return;

    int out_index = ix+iy*w;
    out[out_index] += ALPHA * src[x+y*w];
}


extern "C" void augment_backward_gpu(int w, int h, float ALPHA, float *src, float *dest, float angle, int flip, float scale)
{
    float radians = (float)(angle)*3.14159265/180.;
    int size = w*h;
    augment_backward_kernel<<<cuda_gridsize(size), BLOCK>>>(size, w, h, src, dest, radians, flip, scale, ALPHA);
    check_error(hipPeekAtLastError());
}


__constant__ float c_radians[32];
__constant__ float c_scales[32];
__constant__ int c_flips[32];


__global__ void augment_backward_max_kernel(int size, int w, int h, int out_c, int out_w, int out_h, int gap,
                                            float *src, float *dest, int* indexes)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= size) return;

    int x = id % out_w;
    id /= out_w;
    int y = id % out_h;
    id /= out_h;
    int c = id % out_c;

    int widx = x + y*out_w + c*out_w*out_h;
    int max_i = indexes[widx];
    float dt = src[widx];
    dest[max_i] += dt;
}

__global__ void augment_forward_max_kernel(int size, int w, int h, int c, int out_w, int out_h, int gap,
                                           float *src, float *dest, int* indexes, int n_aug)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= size) return;
    int x = id % out_w;
    id /= out_w;
    int y = id % out_h;
    id /= out_h;
    int ch = id % c;

    int cx = out_w/2;
    int cy = out_h/2;

    int widx = x + y*out_w + ch*out_w*out_h;
    int ridx = x + y*w + ch*w*h;
    int rbase = ch*w*h;
    float max = src[ridx];
    int max_i = ridx;
    int i;
    for (i=0;i<n_aug;i++)
    {
        rbase += gap*out_w + out_w*out_h;
        float rx = c_scales[i]*(cos(c_radians[i])*(x-cx) - sin(c_radians[i])*(y-cy)) + cx;
        float ry = c_scales[i]*(sin(c_radians[i])*(x-cx) + cos(c_radians[i])*(y-cy)) + cy;
        rx = (c_flips[i] ? (w-rx-1) : rx);

        int ix = (int)floorf(rx);
        int iy = (int)floorf(ry);
        if (ix < 0 || ix >= out_w || iy < 0 || iy >= out_h) continue;
        ridx = rbase+ix+iy*out_w;
        //float val = bilinear_interpolate_kernel(src+rbase, out_w, out_h, rx, ry);

        float val = src[ridx];
        max_i = (val > max) ? ridx : max_i;
        max   = (val > max) ? val  : max;
    }

    dest[widx] = max;
    indexes[widx] = max_i;
}


extern "C" void augment_forward_max_gpu(int w, int h, int c, int out_w, int out_h, int gap,
                                        float *src, float *dest, int* indexes,
                                        int n_aug,
                                        float* angles, int* flips, float* scales)
{
    float radians[32];
    float scales_[32];
    int i;
    for (i=0;i<n_aug;i++)
    {
        radians[i] = (float)(angles[i])*3.14159265/180.;
        if (flips[i]==0)
            radians[i] = -radians[i];
        scales_[i] = 1.f / scales[i];
    }

    hipMemcpyToSymbol(HIP_SYMBOL(c_radians), radians, n_aug*sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(c_scales), scales_, n_aug*sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(c_flips), flips, n_aug*sizeof(int));

    int size = out_w*out_h*c;
    augment_forward_max_kernel<<<cuda_gridsize(size), BLOCK>>>(size, w, h, c, out_w, out_h, gap, src, dest, indexes, n_aug);
    check_error(hipPeekAtLastError());
}
extern "C" void augment_backward_max_gpu(int w, int h, int c, int out_w, int out_h, int gap,
                                        float *src, float *dest, int* indexes)
{
    int size = out_w*out_h*c;
    augment_backward_max_kernel<<<cuda_gridsize(size), BLOCK>>>(size, w, h, c, out_w, out_h, gap, src, dest, indexes);
    check_error(hipPeekAtLastError());
}



__global__ void augment_backward_split_kernel(int size, int out_w, int out_h, int out_c,
                                            float *src, float *dest, int* indexes)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= size) return;

    int x = id % out_w;
    id /= out_w;
    int y = id % out_h;
    id /= out_h;
    int c = id % out_c;

    int widx = x + y*out_w + c*out_w*out_h;
    int ridx = indexes[widx];
    float dt = src[widx];
    dest[ridx] += dt;
}

__global__ void augment_forward_split_kernel(int size, int w, int h, int c, int out_w, int out_h, int gap,
                                           float *src, float *dest, int* indexes, int n_aug)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= size) return;
    int x = id % out_w;
    id /= out_w;
    int y = id % out_h;
    id /= out_h;
    int ch = id % (c*(1+n_aug)); // channel in output
    int chf = ch % c; // channel in input
    int augi = ch /c; // augmentation index

    int cx = out_w/2;
    int cy = out_h/2;

    int widx = x + y*out_w + ch*out_w*out_h;
    int rbase = chf*w*h;
    int ridx;

    if (augi==0)
    {
        // augmentation, just copy
        ridx = x + y*w + chf*w*h;
    } else
    {
        rbase += (gap*out_w + out_w*out_h) * augi;
        int i = augi-1;
        float rx = c_scales[i]*(cos(c_radians[i])*(x-cx) - sin(c_radians[i])*(y-cy)) + cx;
        float ry = c_scales[i]*(sin(c_radians[i])*(x-cx) + cos(c_radians[i])*(y-cy)) + cy;
        rx = (c_flips[i] ? (w-rx-1) : rx);

        int ix = (int)floorf(rx);
        int iy = (int)floorf(ry);
        if (ix < 0 || ix >= out_w || iy < 0 || iy >= out_h) return;
        ridx = rbase+ix+iy*out_w;
    }
    float v = src[ridx];
    dest[widx] = v;
    indexes[widx] = ridx;
}


extern "C" void augment_forward_split_gpu(int w, int h, int c, int out_w, int out_h, int gap,
                                        float *src, float *dest, int* indexes,
                                        int n_aug,
                                        float* angles, int* flips, float* scales)
{
    float radians[32];
    float scales_[32];
    int i;
    for (i=0;i<n_aug;i++)
    {
        radians[i] = (float)(angles[i])*3.14159265/180.;
        if (flips[i]==0)
            radians[i] = -radians[i];
        scales_[i] = 1.f / scales[i];
    }
    int out_c = c*(1+n_aug);

    hipMemcpyToSymbol(HIP_SYMBOL(c_radians), radians, n_aug*sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(c_scales), scales_, n_aug*sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(c_flips), flips, n_aug*sizeof(int));

    int size = out_w*out_h*out_c;
    augment_forward_split_kernel<<<cuda_gridsize(size), BLOCK>>>(size, w, h, c, out_w, out_h, gap, src, dest, indexes, n_aug);
    check_error(hipPeekAtLastError());
}
extern "C" void augment_backward_split_gpu(int out_w, int out_h, int out_c,
                                        float *src, float *dest, int* indexes)
{
    int size = out_w*out_h*out_c;
    augment_backward_split_kernel<<<cuda_gridsize(size), BLOCK>>>(size, out_w, out_h, out_c, src, dest, indexes);
    check_error(hipPeekAtLastError());
}

